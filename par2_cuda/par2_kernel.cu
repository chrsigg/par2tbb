#include "hip/hip_runtime.h"
//  This file is part of par2cmdline (a PAR 2.0 compatible file verification and
//  repair tool). See http://parchive.sourceforge.net for details of PAR 2.0.
//
//  GPGPU support using nVidia CUDA technology. Copyright (c) 2008 Vincent Tan.
//  Created 2008-09-20. par2_kernel.cu
//
//  par2cmdline-0.4-tbb is available at http://chuchusoft.com/par2_tbb
//
//  par2cmdline is free software; you can redistribute it and/or modify
//  it under the terms of the GNU General Public License as published by
//  the Free Software Foundation; either version 2 of the License, or
//  (at your option) any later version.
//
//  par2cmdline is distributed in the hope that it will be useful,
//  but WITHOUT ANY WARRANTY; without even the implied warranty of
//  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//  GNU General Public License for more details.
//
//  You should have received a copy of the GNU General Public License
//  along with this program; if not, write to the Free Software
//  Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA  02111-1307  USA

/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

#ifndef _PAR2_KERNEL_H_
  #define _PAR2_KERNEL_H_

  #include <stdio.h>

  #if 1

  __device__
  unsigned
  uc_to_ui(unsigned char i) {
    return i;
  }

  __device__
  unsigned short
  ui_to_us(unsigned i) {
    return i;
  }

  #define TEXREF(N) texRef ## N
  #define TEXFETCH(N, IDX) tex1Dfetch(TEXREF(N), IDX)
  #define LH(N, i) ((unsigned) (TEXFETCH(N, i & 0xFF) ^ TEXFETCH(N, 256U + uc_to_ui(i >> 8))))
  #define LHLH(N, i) LH(N, i) ^ (LH(N, ui_to_us(i >> 16)) << 16)
  #define RSKERNEL(N) \
__global__ \
void \
rsKernel ## N(unsigned* g_odata, const unsigned* g_idata) { \
  const unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x; /* input/output index */ \
  const unsigned data = g_idata[tid]; \
  (unsigned int) atomicXor(g_odata + tid, LHLH(N, data)); \
}

texture<unsigned short, 1, hipReadModeElementType> texRef0;
texture<unsigned short, 1, hipReadModeElementType> texRef1;
texture<unsigned short, 1, hipReadModeElementType> texRef2;
texture<unsigned short, 1, hipReadModeElementType> texRef3;

RSKERNEL(0)
RSKERNEL(1)
RSKERNEL(2)
RSKERNEL(3)

  #else

texture<unsigned short, 1, hipReadModeElementType> texRef0;

__device__
unsigned
lh(unsigned short i) {
  return (unsigned) (tex1Dfetch(texRef0, i & 0xFF) ^ tex1Dfetch(texRef0, 256U + ((unsigned) i >> 8)));
}

__device__
unsigned
lhlh(unsigned i) {
  return lh(i) ^ (lh(i >> 16) << 16);
}

__global__
void
rsKernel0(unsigned* g_odata, const unsigned* g_idata) {
  const unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x; // input/output index
  (unsigned int) atomicXor(g_odata + tid, lhlh(g_idata[tid]));
}

  #endif

#endif // #ifndef _PAR2_KERNEL_H_
