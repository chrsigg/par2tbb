//  This file is part of par2cmdline (a PAR 2.0 compatible file verification and
//  repair tool). See http://parchive.sourceforge.net for details of PAR 2.0.
//
//  GPGPU support using nVidia CUDA technology. Copyright (c) 2008 Vincent Tan.
//  Created 2008-09-20. par2.cu
//
//  par2cmdline-0.4-tbb is available at http://chuchusoft.com/par2_tbb
//
//  par2cmdline is free software; you can redistribute it and/or modify
//  it under the terms of the GNU General Public License as published by
//  the Free Software Foundation; either version 2 of the License, or
//  (at your option) any later version.
//
//  par2cmdline is distributed in the hope that it will be useful,
//  but WITHOUT ANY WARRANTY; without even the implied warranty of
//  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//  GNU General Public License for more details.
//
//  You should have received a copy of the GNU General Public License
//  along with this program; if not, write to the Free Software
//  Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA  02111-1307  USA

/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>
#include <vector>

#define MULTIPLE_STREAMS 1

#include "par2_kernel.cu"

namespace cuda {
namespace internal {
  extern size_t CountDevices(void);
  extern bool SelectDevice(size_t i);

  extern size_t StreamCount(void);

  extern size_t AllocStream(void);
  extern bool DeallocStream(size_t streamidx);
  extern bool SyncStream(size_t streamidx);
  extern bool QueryStream(size_t streamidx);

  extern void* AllocHostMemory(size_t sz);
  extern bool DeallocHostMemory(void* ptr);
  extern void* AllocDeviceMemory(size_t sz); // returns devptr
  extern bool DeallocDeviceMemory(void* devptr);
  extern bool CopyFromHostToDeviceMemory(void* devptr, const void* hostptr, size_t sz, size_t streamidx);
  extern bool CopyFromDeviceToHostMemory(void* hostptr, const void* devptr, size_t sz, size_t streamidx);

  extern bool BindTextureToDeviceMemory(size_t streamidx, void* tex_dev_ptr, size_t tex_size);
  extern bool ProcessViaGPU(size_t streamidx, size_t len, // The number of u32's to process
                            unsigned* dev_obuf, const unsigned* inputbuffer, const unsigned short* host_lh,
                            unsigned* dev_ibuf, unsigned short* dev_lh);

  // up to 4 GPU streams are supported but testing on an 8600M GT showed no improvement above 2:
  enum { NUM_GPU_PROCESSORS = MULTIPLE_STREAMS ? 2 : 1 };

  static hipStream_t streams_[NUM_GPU_PROCESSORS] = {0};
  static size_t streams_size_ = 0;

size_t
CountDevices(void) {
  int count = 0;
  hipError_t err = ::hipGetDeviceCount(&count);
  if (hipSuccess != err)
    return 0;
  return (size_t) count;
}

bool
SelectDevice(size_t i) {
  hipDeviceProp_t deviceProp;
  hipError_t err = ::hipGetDeviceProperties(&deviceProp, (int) i);
//printf("hipGetDeviceProperties -> %d, deviceProp.major = %d\n", err, deviceProp.major);
  if (deviceProp.major < 1)
    return false;
  err = ::hipSetDevice((int) i);
//if (err) printf("hipSetDevice(%d) -> %d\n", i, err);
  if (hipSuccess != err)
    return false;

  err = ::hipDeviceSynchronize();
//if (err) printf("hipDeviceSynchronize -> %d\n", err);
  return true;//hipSuccess == err;
}

size_t
StreamCount(void) {
  return NUM_GPU_PROCESSORS;
}

size_t
AllocStream(void) {
  if (streams_size_ == NUM_GPU_PROCESSORS)
    return 0; // no room left

#if !defined(WIN32)
  assert(streams_size_ < NUM_GPU_PROCESSORS);
  assert(0 == streams_[streams_size_]);
#endif

  hipStream_t stream = 0;
  hipError_t err = ::hipStreamCreate(&stream);
  bool ok = hipSuccess == err;
  if (ok) {
    streams_[streams_size_++] = stream;
  }
  return ok ? streams_size_ : 0;
}

bool
DeallocStream(size_t streamidx) {
  hipError_t err = 0 < streamidx && streamidx <= streams_size_ ? ::hipStreamDestroy(streams_[streamidx-1]) : hipErrorInvalidHandle;
  bool res = hipSuccess == err;
  if (res)
    streams_[streamidx-1] = 0;
  return res;
}

bool
SyncStream(size_t streamidx) {
  hipError_t err;
  if (streamidx)
    err = 0 < streamidx && streamidx <= streams_size_ ? ::hipStreamSynchronize(streams_[streamidx-1]) : hipErrorInvalidHandle;
  else
    err = ::hipDeviceSynchronize();

//if (err) printf("hipStreamSynchronize(%d) -> %d (%s)\n", streamidx, err, hipGetErrorString(err));
  return hipSuccess == err;
}

bool
QueryStream(size_t streamidx) {
  hipError_t err = 0 < streamidx && streamidx <= streams_size_ ? ::hipStreamQuery(streams_[streamidx-1]) : hipErrorInvalidHandle;
//if (!(hipSuccess == err || hipErrorNotReady == err)) printf("hipStreamQuery(%d) -> %d (%s)\n", streamidx, err, hipGetErrorString(err));
#if !defined(WIN32)
  assert(hipSuccess == err || hipErrorNotReady == err);
#endif
  return hipSuccess == err;
}

void*
AllocHostMemory(size_t sz) {
  void* ptr = NULL;
  hipError_t err = ::hipHostMalloc(&ptr, sz);
//if (err) printf("hipHostMalloc(%u) -> %d (%s) %p\n", (unsigned) sz, err, hipGetErrorString(err), ptr);
  return hipSuccess == err ? ptr : NULL;
}

bool
DeallocHostMemory(void* ptr) {
  hipError_t err = ::hipHostFree(ptr);
//printf("hipHostFree(%p) -> %d (%s)\n", ptr, err, hipGetErrorString(err));
  return hipSuccess == err;
}

void*
AllocDeviceMemory(size_t sz) {
  void* devptr = NULL;
  hipError_t err = ::hipMalloc(&devptr, sz);
//if (err) printf("hipMalloc(%u) -> %d (%s) %p\n", (unsigned) sz, err, hipGetErrorString(err), devptr);
  return hipSuccess == err ? devptr : NULL;
}

bool
DeallocDeviceMemory(void* devptr) {
  hipError_t err = ::hipFree(devptr);
//printf("hipFree(%p) -> %d (%s)\n", devptr, err, hipGetErrorString(err));
  return hipSuccess == err;
}

bool
CopyFromHostToDeviceMemory(void* devptr, const void* hostptr, size_t sz, size_t streamidx) {
  hipError_t err;
  if (streamidx)
    err = 0 < streamidx && streamidx <= streams_size_
          ? ::hipMemcpyAsync(devptr, hostptr, sz, hipMemcpyHostToDevice, streams_[streamidx-1])
          : hipErrorInvalidHandle;
  else
    err = ::hipMemcpy(devptr, hostptr, sz, hipMemcpyHostToDevice);

//if (err) printf("hipMemcpyHostToDevice(%p, %p, %u, %d) -> %d (%s)\n", devptr, hostptr, (unsigned) sz, streamidx, err, hipGetErrorString(err));
  return hipSuccess == err;
}

bool
CopyFromDeviceToHostMemory(void* hostptr, const void* devptr, size_t sz, size_t streamidx) {
  hipError_t err;
  if (streamidx)
    err = 0 < streamidx && streamidx <= streams_size_
          ? ::hipMemcpyAsync(hostptr, devptr, sz, hipMemcpyDeviceToHost, streams_[streamidx-1])
          : hipErrorInvalidHandle;
  else
    err = ::hipMemcpy(hostptr, devptr, sz, hipMemcpyDeviceToHost);

//if (err) printf("hipMemcpyDeviceToHost(%p, %p, %u, %d) -> %d (%s)\n", devptr, hostptr, (unsigned) sz, streamidx, err, hipGetErrorString(err));
  return hipSuccess == err;
}

bool BindTextureToDeviceMemory(size_t streamidx, void* tex_dev_ptr, size_t tex_size) {
  // texRef MUST be global and should be declared in the *_kernel.cu file
  hipError_t err;

  if (1 == streamidx)
    err = ::hipBindTexture(NULL, texRef0, tex_dev_ptr, tex_size);
#if MULTIPLE_STREAMS
  else if (2 == streamidx)
    err = ::hipBindTexture(NULL, texRef1, tex_dev_ptr, tex_size);
  else if (3 == streamidx)
    err = ::hipBindTexture(NULL, texRef2, tex_dev_ptr, tex_size);
  else if (4 == streamidx)
    err = ::hipBindTexture(NULL, texRef3, tex_dev_ptr, tex_size);
#endif
  else
    err = hipErrorInvalidHandle;
  return hipSuccess == err;
}

bool
ProcessViaGPU(size_t streamidx, size_t len, // The number of u32's to process
              unsigned* dev_obuf, const unsigned* inputbuffer, const unsigned short* host_lh,
              unsigned* dev_ibuf, unsigned short* dev_lh) {
//printf("ProcessViaGPU(streamidx=%lu, len=%lx, dev_obuf=%p, inputbuffer=%p, host_lh=%p, dev_ibuf=%p, dev_lh=%p)\n", streamidx, len, dev_obuf, inputbuffer, host_lh, dev_ibuf, dev_lh);

  //assert(0 == (len & 127));
  if (0 == streamidx || streamidx > streams_size_)
    return false;

  const size_t mem_size = sizeof(unsigned) * len;
  //hipError_t err;

  if (!CopyFromHostToDeviceMemory(dev_ibuf, inputbuffer, mem_size, streamidx))
    return false;

  if (!SyncStream(streamidx))
    return false;

  if (!CopyFromHostToDeviceMemory(dev_lh, host_lh, 512 * sizeof(unsigned short), streamidx))
    return false;

  //  NVIDIA recommend at least 64 threads so let's use 128 to ensure
  //  that the GPU is kept occupied. Then the number of blocks, b, is
  //  determined by the number, n, of uint32's that are to be processed
  //  (since the GPU operates using 32-bit registers as per section
  //  3.1 of the CUDA programming guide):
  //    b = n / 128
  //  For example, if n = 128K then b = 128K / 128 = 1K. So:
  //    dim3  grid( 1024, 1, 1);
  //    dim3  threads( 128 /* num_threads */, 1, 1);
  //    rsKernel<<< grid, threads, 0 /*mem_size*/, streamid >>>( dev_ibuf );
  //
  //  If the number of uint32 values is not evenly divisible by 128 then
  //  process the last block as 128 values but only copy back the remaining
  //  n mod 128 values.

  dim3 grid( unsigned(127+len)/128, 1, 1);
  dim3 threads( 128 /* num_threads */, 1, 1);

  // enqueue request to execute the kernel
  if (1 == streamidx)
    rsKernel0<<< grid, threads, 0 /*mem_size*/, streams_[streamidx-1] >>>( dev_obuf, dev_ibuf );
#if MULTIPLE_STREAMS
  else if (2 == streamidx)
    rsKernel1<<< grid, threads, 0 /*mem_size*/, streams_[streamidx-1] >>>( dev_obuf, dev_ibuf );
  else if (3 == streamidx)
    rsKernel2<<< grid, threads, 0 /*mem_size*/, streams_[streamidx-1] >>>( dev_obuf, dev_ibuf );
  else if (4 == streamidx)
    rsKernel3<<< grid, threads, 0 /*mem_size*/, streams_[streamidx-1] >>>( dev_obuf, dev_ibuf );
#endif
  else
    return false;

  return true;
}

} // namespace internal
} // namespace cuda

